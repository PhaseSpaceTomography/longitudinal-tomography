#include "hip/hip_runtime.h"
/**
 * @file reconstruct.cu
 *
 * @author Bernardo Abreu Figueiredo
 * Contact: bernardo.abreu.figueiredo@cern.ch
 *
 * CUDA kernels that handle phase space reconstruction for single precision floating-point numbers.
 */

#include <cub/block/block_reduce.cuh>

// Back projection using flattened arrays and a block-wide reduction.
// Implementation with fixed block_size and items_per_array, but variable number of profiles for the reduction
// Must be called with block size 32.
extern "C"
__global__ void back_project(float * __restrict__ weights,                     // inn/out
                             int * __restrict__ flat_points,                    // inn
                             const float * __restrict__ flat_profiles,         // inn
                             const int npart, const int nprof) {                // inn
    const int BLOCK_SIZE = 32;
    const int ITEMS_PER_ARRAY = 16;
    const int ITEMS_PER_IT = BLOCK_SIZE * ITEMS_PER_ARRAY;
    int iterations = (nprof + ITEMS_PER_IT - 1) / ITEMS_PER_IT;

    float aggregate = 0.0f;

    for(int i = 0; i < iterations; i++)
    {
        typedef hipcub::BlockReduce<float, BLOCK_SIZE> BlockReduce;

        // allocate shared memory for BlockReduce
        __shared__ typename BlockReduce::TempStorage temp_storage;

        float weight_prof[ITEMS_PER_ARRAY];

        for (int j = 0; j < ITEMS_PER_ARRAY; j++)
        {
            int index = i * ITEMS_PER_IT + j * blockDim.x + threadIdx.x;
            if (index < nprof)
                weight_prof[j] = flat_profiles[flat_points[blockIdx.x * nprof + index]];
            else
                weight_prof[j] = 0.0f;
        }

        __syncthreads();

        aggregate += BlockReduce(temp_storage).Sum(weight_prof);
    }

    if (threadIdx.x == 0)
        weights[blockIdx.x] += aggregate;
}

// Projection using flattened arrays.
// This function does not iterate, so the
// amount of threads should be at least equal to the product of npart and nprof.
extern "C"
__global__ void project(float * __restrict__ flat_rec,         // inn/out
                        const int * __restrict__ flat_points,   // inn
                        const float * __restrict__ weights,    // inn
                        const int npart, const int nprof) {     // inn
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid < npart * nprof)
    {
        int idx = flat_points[tid];
        atomicAdd(&flat_rec[idx], weights[tid / nprof]);
    }
}

// Array clipping function to set values below a threshold
// to the respective value.
// This function does not iterate, so the
// amount of threads should be at least equal to the length.
extern "C"
__global__ void clip(float *array, // inn/out
          const int length,
          const double clip_val) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid < length)
    {
        if (array[tid] < (float) clip_val)
            array[tid] = (float) clip_val;
    }
}

// Calculates the difference between the reconstructed profile
// and the flat profiles.
// This function iterates, however to reduce multiple iterations,
// the amount of threads should be at least equal to all_bins if possible.
extern "C"
__global__ void find_difference_profile(float * __restrict__ diff_prof,    // out
                             const float * __restrict__ flat_rec,          // inn
                             const float * __restrict__ flat_profiles,     // inn
                             const int all_bins) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    for (int i = tid; i < all_bins; i += blockDim.x * gridDim.x)
        if (i < all_bins)
            diff_prof[i] = flat_profiles[i] - flat_rec[i];
}

// Multiplies the profile differences with the reciprocal particle array
// to compensate for the amount of particles.
// This function does not iterate, so the
// amount of threads should be at least equal to the product of nprof and nbins.
extern "C"
__global__ void compensate_particle_amount(float * __restrict__ diff_prof,     // inn/out
                                const float * __restrict__ rparts,             // inn
                                const int nprof,
                                const int nbins) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid < nprof * nbins) {
        diff_prof[tid] *= rparts[tid];
    }
}

// Counts the particles in each bin.
// This function does not iterate, so the
// amount of threads should be at least equal to the product of npart and nprof.
extern "C"
__global__ void count_particles_in_bin(float * __restrict__ rparts,    // out
                            const int * __restrict__ xp,                // inn
                            const int nprof,
                            const int npart,
                            const int nbins) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid < npart * nprof)
    {
        int j = tid % nprof;
        int bin = xp[tid];
        atomicAdd(&rparts[j * nbins + bin], 1);
    }
}

// Calculates the reciprocal of the counted particles per bin.
// This function does not iterate, so the
// amount of threads should be at least equal to the product of nprof and nbins.
extern "C"
__global__ void calculate_reciprocal(float *rparts,   // inn/out
                          const int nbins,
                          const int nprof,
                          const double maxVal) {
    const int all_bins = nprof * nbins;

    // Setting 0's to 1's to avoid zero division and creating reciprocal
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid < all_bins) {
        if (rparts[tid] == 0.0f)
            rparts[tid] = 1.0f;
        rparts[tid] = (float) maxVal / rparts[tid];
    }
}

// Creates a flattened representation of the particle coordinates
// used for indexing. 
// This function does not iterate, so the
// amount of threads should be at least equal to the product of npart and nprof.
extern "C"
__global__ void create_flat_points(int *flat_points,    // inn/out
                        const int npart,
                        const int nprof,
                        const int nbins) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid < npart * nprof)
        flat_points[tid] += nbins * (tid % nprof);
}