/**
 * @file reconstruct.cu
 *
 * @author Bernardo Abreu Figueiredo
 * Contact: bernardo.abreu.figueiredo@cern.ch
 *
 * CUDA kernels that handle phase space reconstruction.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

// Back projection using flattened arrays
extern "C"
__global__ void back_project(double *weights,                     // inn/out
                             int *flat_points,                    // inn
                             const double *flat_profiles,         // inn
                             const int npart, const int nprof) {  // inn
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    
    if(tid < npart * nprof)
    {
        int idx = flat_points[tid];
        atomicAdd(&weights[tid / nprof], flat_profiles[idx]);
    }
}

// Projections using flattened arrays
extern "C"
__global__ void project(double *flat_rec,                       // inn/out
                        int *flat_points,                       // inn
                        const double *weights,                  // inn
                        const int npart, const int nprof) {     // inn
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid < npart * nprof)
    {
        int idx = flat_points[tid];
        atomicAdd(&flat_rec[idx], weights[tid / nprof]);
    }
}

// extern "C"
// __global__ void normalize(double *flat_rec, // inn/out
//                const int nprof,
//                const int nbins) {
//     // TODO
//     double sum_waterfall = 0.0;
// #pragma omp parallel for reduction(+ : sum_waterfall)
//     for (int i = 0; i < nprof; i++) {
//         double sum_profile = 0;
//         for (int j = 0; j < nbins; j++)
//             sum_profile += flat_rec[i * nbins + j];
//         for (int j = 0; j < nbins; j++)
//             flat_rec[i * nbins + j] /= sum_profile;
//         sum_waterfall += sum_profile;
//     }

//     if (sum_waterfall <= 0)
//         throw std::runtime_error("Phase space reduced to zeroes!");
// }

extern "C"
__global__ void clip(double *array, // inn/out
          const int length,
          const double clip_val) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid < length)
    {
        if (array[tid] < clip_val)
            array[tid] = clip_val;
    }
}


extern "C"
__global__ void find_difference_profile(double *diff_prof,           // out
                             const double *flat_rec,      // inn
                             const double *flat_profiles, // inn
                             const int all_bins) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    for (int i = tid; i < all_bins; i += blockDim.x * gridDim.x)
        if (i < all_bins)
            diff_prof[i] = flat_profiles[i] - flat_rec[i];
}

// extern "C"
// __global__ double discrepancy(const double *diff_prof,   // inn
//                    const int nprof,
//                    const int nbins) {
//     int all_bins = nprof * nbins;
//     double squared_sum = 0;

//     int tid = threadIdx.x + blockDim.x * blockIdx.x;
//     for (int i = tid; i < all_bins; i += blockDim.x * gridDim.x) {
//         squared_sum += pow(diff_prof[i], 2.0);
//     }

//     return sqrt(squared_sum / (nprof * nbins));
// }

extern "C"
__global__ void compensate_particle_amount(double *diff_prof,        // inn/out
                                double *rparts,          // inn
                                const int nprof,
                                const int nbins) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid < nprof * nbins) {
        diff_prof[tid] *= rparts[tid];
    }
}

// // Parallel reduction?
// extern "C"
// __global__ double max_2d(double **arr,  // inn
//               const int x_axis,
//               const int y_axis) {
//     double max_bin_val = 0;
//     for (int i = 0; i < y_axis; i++)
//         for (int j = 0; j < x_axis; j++)
//             if (max_bin_val < arr[i][j])
//                 max_bin_val = arr[i][j];
//     return max_bin_val;
// }

// // Parallel reduction?
// extern "C"
// __global__ double max_1d(double *arr, const int length) {
//     double max_bin_val = 0;
//     for (int i = 0; i < length; i++)
//         if (max_bin_val < arr[i])
//             max_bin_val = arr[i];
//     return max_bin_val;
// }

// Atomic add?
extern "C"
__global__ void count_particles_in_bin(double *rparts,      // out
                            const int *xp,       // inn
                            const int nprof,
                            const int npart,
                            const int nbins) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    for (int i = tid; i < npart * nprof; i += blockDim.x * gridDim.x) {
        if (i < npart * nprof) {
            int j = i % nprof;
            int bin = xp[i];
            atomicAdd(&rparts[j * nbins + bin], 1);
        }
    }
}


extern "C"
__global__ void calculate_reciprocal(double *rparts,   // inn/out
                          const int nbins,
                          const int nprof,
                          const double maxVal) {
    const int all_bins = nprof * nbins;

    // Setting 0's to 1's to avoid zero division and creating reciprocal
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid < all_bins) {
        if (rparts[tid] == 0.0)
            rparts[tid] = 1.0;
        rparts[tid] = maxVal / rparts[tid];
    }
}

extern "C"
__global__ void create_flat_points(int *flat_points,    // inn/out
                        const int npart,
                        const int nprof,
                        const int nbins) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid < npart)
        for (int j = 0; j < nprof; j++)
            flat_points[tid * nprof + j] += nbins * j;
}