
#include <hip/hip_runtime.h>
/**
 * @file kick_and_drift.cu
 *
 * @author Bernardo Abreu Figueiredo
 * Contact: bernardo.abreu.figueiredo@cern.ch
 *
 * CUDA kernels that handles particle tracking (kicking and
 * drifting).
 */


extern "C"
__global__ void kick_up(const double *dphi,
                        double *denergy,
                        const double rfv1,
                        const double rfv2,
                        const double phi0,
                        const double phi12,
                        const double hratio,
                        const int nr_particles,
                        const double acc_kick) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    for (int i = tid; i < nr_particles; i += blockDim.x * gridDim.x)
        denergy[i] += rfv1 * sin(dphi[i] + phi0)
                      + rfv2 * sin(hratio * (dphi[i] + phi0 - phi12)) - acc_kick;
}

extern "C"
__global__ void kick_down(const double *dphi,
                          double *denergy,
                          const double rfv1,
                          const double rfv2,
                          const double phi0,
                          const double phi12,
                          const double hratio,
                          const int nr_particles,
                          const double acc_kick) {

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    for (int i = tid; i < nr_particles; i += blockDim.x * gridDim.x)
        denergy[i] -= rfv1 * sin(dphi[i] + phi0)
                      + rfv2 * sin(hratio * (dphi[i] + phi0 - phi12)) - acc_kick;
}

extern "C"
__global__ void drift_up(double *dphi,
                         const double *denergy,
                         const double drift_coef,
                         const int nr_particles) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    for(int i = tid; i < nr_particles; i += blockDim.x * gridDim.x) {
        dphi[i] -= drift_coef * denergy[i];
    }
}

extern "C"
__global__ void drift_down(double *dphi,
                         const double *denergy,
                         const double drift_coef,
                         const int nr_particles) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    for(int i = tid; i < nr_particles; i += blockDim.x * gridDim.x) {
        dphi[i] += drift_coef * denergy[i];
    }
}

extern "C"
__global__ void kick_drift_up_simultaneously(double *dphi,
                         double *denergy,
                         const double drift_coef,
                         const double rfv1,
                         const double rfv2,
                         const double phi0,
                         const double phi12,
                         const double hratio,
                         const int nr_particles,
                         const double acc_kick) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    for(int i = tid; i < nr_particles; i += blockDim.x * gridDim.x) {
        dphi[i] -= drift_coef * denergy[i];
        denergy[i] += rfv1 * sin(dphi[i] + phi0)
                      + rfv2 * sin(hratio * (dphi[i] + phi0 - phi12)) - acc_kick;
    }
}

extern "C"
__global__ void kick_drift_down_simultaneously(double *dphi,
                         double *denergy,
                         const double drift_coef,
                         const double rfv1,
                         const double rfv2,
                         const double phi0,
                         const double phi12,
                         const double hratio,
                         const int nr_particles,
                         const double acc_kick) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    for(int i = tid; i < nr_particles; i += blockDim.x * gridDim.x) {
        denergy[i] -= rfv1 * sin(dphi[i] + phi0)
                      + rfv2 * sin(hratio * (dphi[i] + phi0 - phi12)) - acc_kick;
        dphi[i] += drift_coef * denergy[i];
    }
}